#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of gSLICr
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_GPU.h"
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_cluster_base.h"
/*-------------------------------------------------------------------------*/
using namespace std;
using namespace gSLICr;
using namespace gSLICr::objects;
using namespace gSLICr::engines;
/*-------------------------------------------------------------------------*/
/**
*kernel function defines
*/
/*-------------------------------------------------------------------------*/
__global__ void Find_Link_Matrix(const Vector4u* inimg, Vector4f* outimg, Vector2i img_size);
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
__global__ void Find_Link_Matrix(const Vector4u * inimg, Vector4f * outimg, Vector2i img_size)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > img_size.x - 1 || y > img_size.y - 1) return;

	find_link_matrix_base(inimg, outimg, img_size, x, y);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
