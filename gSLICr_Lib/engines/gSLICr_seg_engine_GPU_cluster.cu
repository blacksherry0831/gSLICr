#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of gSLICr
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_GPU.h"
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_cluster_base.h"
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_GPU_cluster.h"
/*-------------------------------------------------------------------------*/
using namespace gSLICr;
using namespace gSLICr::objects;
using namespace gSLICr::engines;
/*-------------------------------------------------------------------------*/
#include <vector>
/*-------------------------------------------------------------------------*/
/**
*kernel function defines
*/
/*-------------------------------------------------------------------------*/
__global__ void Kernel_Find_Adjacency_Matrix(
	const int * inimg,
	int * outimg,
	const  gSLICr::Vector2i img_size,
	const  gSLICr::Vector2i adj_size)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	const int BORDER = 1;
	if (x < img_size.x - BORDER &&
		y < img_size.y - BORDER &&
		x >= BORDER &&
		y >= BORDER) {

		find_adjacency_matrix_base(inimg, outimg, img_size, adj_size, x, y);

	}
		
}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
__global__ void Kernel_Cvt_Spixel_to_LThetaM(
	const spixel_info* _spixel_list_src,
	spixel_info* _spixel_list_dst,
	const Vector2i _map_size)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > _map_size.x - 1 || y > _map_size.y - 1) return;
	
	cvt_spixel_to_l_theta_m_raw(
		_spixel_list_src,
		_spixel_list_dst,
		_map_size,
		x,
		y);

}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
__global__ void Kernel_Cvt_Spixel_Similar(
	const int *				_adj_img,
	float *					_similar_img,
	const  spixel_info*		_spixel_list,
	const  gSLICr::Vector2i _adj_size,
	const float _L_Color_th,
	const float _M_Color_th,
	const float _Theta_Color_th,
	const float _L_Gray_th,
	const float _M_Gray_th,
	const float _Theta_Gray_th,
	const float _M_Gray_Color_th)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > _adj_size.x - 1 || y > _adj_size.y - 1) return;

	cvt_spixel_similar(
				_adj_img,
				_similar_img,
				_spixel_list,
				_adj_size,
				x,
				y,
				_L_Color_th,
				_M_Color_th,
				_Theta_Color_th,
				_L_Gray_th,
				_M_Gray_th,
				_Theta_Gray_th,
				_M_Gray_Color_th);

}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
seg_engine_GPU_cluster::seg_engine_GPU_cluster(const objects::settings& in_settings) : seg_engine_GPU(in_settings)
{

	this->malloc_Spixel_Mem();

	cluster_idx_img = new IntImage(gSLICr_settings.img_size, true, true);
	m_spixel_map_cvt = new SpixelMap(spixel_map_size, true, true);
	
}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
seg_engine_GPU_cluster::~seg_engine_GPU_cluster()
{
	this->free_Spixel_Mem();

	if (cluster_idx_img  != NULL)				delete cluster_idx_img;
	if (m_spixel_map_cvt != NULL)				delete m_spixel_map_cvt;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::malloc_Spixel_Mem()
{
	const int AdjDim = this->SpixelNum();
	const Vector2i link_size_t(AdjDim, AdjDim);
	const Vector2i map_size_t(AdjDim, 1);

	adj_img = new IntImage(link_size_t, true, true);
	similar_img = new FloatImage(link_size_t, true, true);
	cluster_map_img = new IntImage(map_size_t, true, true);

}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::free_Spixel_Mem()
{
	if (adj_img != NULL)				delete adj_img;
	if (similar_img != NULL)			delete similar_img;
	if (cluster_map_img != NULL)		delete cluster_map_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::SetClusterLThetaM_Threshold(
	const float _L_Color_th,
	const float _M_Color_th,
	const float _Theta_Color_th,
	const float _L_Gray_th,
	const float _M_Gray_th,
	const float _Theta_Gray_th,
	const float _M_Gray_Color_th)
{
	mClusterL_Color_Threshold= _L_Color_th;
	mClusterM_Color_Threshold= _M_Color_th;
	mClusterTheta_Color_Threshold= _Theta_Color_th;

	mClusterL_Gray_Threshold= _L_Gray_th;
	mClusterM_Gray_Threshold= _M_Gray_th;
	mClusterTheta_Gray_Threshold= _Theta_Gray_th;

	mClusterM_Gray_Color_Threshold= _M_Gray_Color_th;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const dim3 gSLICr::engines::seg_engine_GPU_cluster::GetSuperPixelGridDim() const
{
	const Vector2i img_size = spixel_map->noDims;
	return GetGrid2Dim(img_size);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const dim3 gSLICr::engines::seg_engine_GPU_cluster::GetPixelGridDim() const
{
	const Vector2i img_size = source_img->noDims;
	return GetGrid2Dim(img_size);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const dim3 gSLICr::engines::seg_engine_GPU_cluster::GetGrid2Dim(Vector2i _size)  const
{
	const dim3 blockSize= GetBlockDim();
	const dim3 gridSize((int)ceil((float)_size.x / (float)blockSize.x), (int)ceil((float)_size.y / (float)blockSize.y));
	return  gridSize;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const dim3 gSLICr::engines::seg_engine_GPU_cluster::GetBlockDim() const
{
	const dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
	return blockSize;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Find_Adjacency_Matrix_E()
{
	const int* inimg_ptr = idx_img->GetData(MEMORYDEVICE_CUDA);
	int* outimg_ptr = adj_img->GetData(MEMORYDEVICE_CUDA);
	const Vector2i img_size = source_img->noDims;
	const Vector2i adj_size = adj_img->noDims;

	const dim3 blockSize	=	GetBlockDim();
	const dim3 gridSize		=	GetPixelGridDim();

	Kernel_Find_Adjacency_Matrix << <gridSize, blockSize >> >(inimg_ptr, outimg_ptr, img_size,adj_size);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Cvt_Spixel_Similar()
{
	const int*	adj_mat_ptr = adj_img->GetData(MEMORYDEVICE_CUDA);
	float*	similar_mat_ptr =similar_img->GetData(MEMORYDEVICE_CUDA);
	spixel_info* spixel_list_cvt = m_spixel_map_cvt->GetData(MEMORYDEVICE_CUDA);

	const Vector2i map_adj_size = adj_img->noDims;

	const dim3 blockSize	=	GetBlockDim();
	const dim3 gridSize		=	GetGrid2Dim(map_adj_size);

	Kernel_Cvt_Spixel_Similar << <gridSize, blockSize >> > (
		adj_mat_ptr,
		similar_mat_ptr,
		spixel_list_cvt,
		map_adj_size,
		mClusterL_Color_Threshold,
		mClusterM_Color_Threshold,
		mClusterTheta_Color_Threshold,
		mClusterL_Gray_Threshold,
		mClusterM_Gray_Threshold,
		mClusterTheta_Gray_Threshold,
		mClusterM_Gray_Color_Threshold);

}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Cvt_Spixel_to_LThetaM()
{

	assert(gSLICr_settings.color_space == COLOR_SPACE::CIELAB);

	const spixel_info* spixel_list		=	spixel_map->GetData(MEMORYDEVICE_CUDA);
		  spixel_info* spixel_list_cvt	=	m_spixel_map_cvt->GetData(MEMORYDEVICE_CUDA);
	const Vector2i map_size = spixel_map->noDims;
	
	const dim3 blockSize = GetBlockDim();
	const dim3 gridSize =  GetSuperPixelGridDim();

	Kernel_Cvt_Spixel_to_LThetaM << <gridSize, blockSize >> > (spixel_list, spixel_list_cvt, map_size);
	
}
/*-----------------------------------------*/
/**
*
*/
/*-----------------------------------------*/
void gSLICr::engines::getMatTriangular_Float(float* _mat, const int _wh)
{

	for (size_t ri = 0; ri <_wh; ri++) {
		for (size_t ci = 0; ci < ri; ci++) {
			const int IDX = ri*_wh + ci;
			_mat[IDX] = 0;
		}
	}

}
/*-----------------------------------------*/
/**
*
*/
/*-----------------------------------------*/
bool gSLICr::engines::RemoveLine_Float(
	float* _mat,
	const int _wh,
	const int _li)
{

	for (size_t ci = 0; ci < _wh; ci++) {
		const int IDX_I = _li*_wh + ci;
		const int I_v = _mat[IDX_I];
		_mat[IDX_I] = 0;
	}

	return false;
}
/*-----------------------------------------*/
/**
*
*/
/*-----------------------------------------*/
bool gSLICr::engines::MergeNeighbor_Float(
	float * _mat,
	const int _wh,
	const int _i,
	const int _j)
{

	for (size_t ci = 0; ci < _wh; ci++) {
		const int IDX_I = _i* _wh + ci;
		const int IDX_J = _j* _wh + ci;
		const int I_v = _mat[IDX_I];
		const int J_v = _mat[IDX_J];
		const int IJ_v = I_v | J_v;
		_mat[IDX_J] = IJ_v;
	}

	return true;
}
/*-----------------------------------------*/
/**
*
*/
/*-----------------------------------------*/
void gSLICr::engines::getMatCluster_Float(float * _mat, const int _wh)
{
	for (int ci = _wh - 1; ci >= 0; ci--) {

		std::vector<int> sameClass;

		for (int ri = 0; ri <_wh; ri++) {
			const int IDX = ci + ri*_wh;
			const int W = _mat[IDX];
			if (W) {
				sameClass.push_back(ri);
			}
		}

		if (sameClass.size() > 0) {
			const int ri_min = sameClass.at(0);
			for (int ri = 1; ri < sameClass.size(); ri++) {
				const int ri_current = sameClass.at(ri);
				MergeNeighbor_Float(_mat, _wh, ri_current, ri_min);
				RemoveLine_Float(_mat, _wh, ri_current);
			}
		}
	}
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Cvt_Similar_to_Cluster_CPU()
{
	const FloatImage * similar_img_t= this->Get_Similar_Matrix();
	float* similar_ptr = (float*) similar_img_t->GetData(MEMORYDEVICE_CPU);
	const int WH = similar_img_t->noDims.height;
	assert(similar_img_t->noDims.height == similar_img_t->noDims.width);

	getMatTriangular_Float(similar_ptr,WH);

	getMatCluster_Float(similar_ptr, WH);

	int * map=cluster_map_img->GetData(MEMORYDEVICE_CPU);
	
	getMapCluster_Idx(map,similar_ptr,WH);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Cvt_Labels_to_ClusterLabels_CPU()
{
	const FloatImage * similar_img_t = this->Get_Similar_Matrix();
	const IntImage *labels_idx= Get_Idx();
	const int WIDTH  = labels_idx->noDims.width;
	const int HEIGHT = labels_idx->noDims.height;
	const int WH = similar_img_t->noDims.width;

	const int*  map_ptr = cluster_map_img->GetData(MEMORYDEVICE_CPU);
	const int*  labels_idx_ptr =(int*) labels_idx->GetData(MEMORYDEVICE_CPU);
	int*  cluster_idx_ptr = (int*)cluster_idx_img->GetData(MEMORYDEVICE_CPU);


	for (int ci = 0; ci <WIDTH; ci++){
		for (int ri = 0; ri < HEIGHT; ri++){
			const int IDX = ci + ri*WIDTH;
			const int LABEL_OLD = labels_idx_ptr[IDX];
			const int LABEL_NEW = map_ptr[LABEL_OLD];
			cluster_idx_ptr[IDX] = LABEL_NEW;
		}
	}
	
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::getMapCluster_Idx(int* _map, const float * _mat, const int _wh)
{

	for (int ri = 0; ri < _wh; ri++){

		for (int ci = 0; ci < _wh; ci++){

			const int IDX = ci + ri*_wh;

			const int N = _mat[IDX];

			if (N){
				_map[ci] = ri;
			}


		}

	}
	
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Perform_Cluster()
{
	this->Update_Cluster_Center();
	this->Find_Adjacency_Matrix_E();
	this->Cvt_Spixel_to_LThetaM();
	this->Cvt_Spixel_Similar();

	hipDeviceSynchronize();

}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Perform_Cluster_CPU()
{
	this->Cvt_Similar_to_Cluster_CPU();

	this->Cvt_Labels_to_ClusterLabels_CPU();
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
int gSLICr::engines::seg_engine_GPU_cluster::SpixelNum() const
{
	return this->spixel_map->noDims.x * this->spixel_map->noDims.y;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const IntImage * gSLICr::engines::seg_engine_GPU_cluster::Do_Adjacency_Matrix_Cpy_Dev_to_Host()
{
	adj_img->UpdateHostFromDevice();
	return adj_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const IntImage * gSLICr::engines::seg_engine_GPU_cluster::Get_Adjacency_Matrix()
{
	return adj_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const FloatImage * gSLICr::engines::seg_engine_GPU_cluster::Do_Similar_Matrix_Cpy_Dev_to_Host()
{
	similar_img->UpdateHostFromDevice();
	return similar_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const FloatImage * gSLICr::engines::seg_engine_GPU_cluster::Get_Similar_Matrix()
{
	return similar_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const IntImage * gSLICr::engines::seg_engine_GPU_cluster::Get_Cluster_Idx_Seg_Matrix()
{
	return  cluster_idx_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const IntImage * gSLICr::engines::seg_engine_GPU_cluster::Do_Cluster_Idx_Seg_Matrix_Cpy_Host_to_Dev()
{
	cluster_idx_img->UpdateDeviceFromHost();
	return cluster_idx_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const SpixelMap * gSLICr::engines::seg_engine_GPU_cluster::Do_Spixel_Map_Cvt_Spy_Dev_to_Host()
{
	m_spixel_map_cvt->UpdateHostFromDevice();
	return m_spixel_map_cvt;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const SpixelMap * gSLICr::engines::seg_engine_GPU_cluster::Get_Spixel_Map_Cvt()
{
	return m_spixel_map_cvt;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Draw_Segmentation_Cluster_Result(UChar4Image * _out_img)
{
	this->Draw_Segmentation_Result_Ex(source_img, cluster_idx_img,_out_img);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/

/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/