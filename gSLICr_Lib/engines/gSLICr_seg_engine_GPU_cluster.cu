#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of gSLICr
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_GPU.h"
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_cluster_base.h"
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_GPU_cluster.h"
/*-------------------------------------------------------------------------*/
using namespace gSLICr;
using namespace gSLICr::objects;
using namespace gSLICr::engines;
/*-------------------------------------------------------------------------*/
/**
*kernel function defines
*/
/*-------------------------------------------------------------------------*/
__global__ void Kernel_Find_Adjacency_Matrix(
	const int * inimg,
	int * outimg,
	const  gSLICr::Vector2i img_size,
	const  gSLICr::Vector2i adj_size)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	const int BORDER = 1;
	if (x < img_size.x - BORDER &&
		y < img_size.y - BORDER &&
		x >= BORDER &&
		y >= BORDER) {

		find_adjacency_matrix_base(inimg, outimg, img_size, adj_size, x, y);

	}
		
}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
__global__ void Kernel_Cvt_Spixel_to_LThetaM(
	const spixel_info* _spixel_list_src,
	spixel_info* _spixel_list_dst,
	const Vector2i _map_size)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > _map_size.x - 1 || y > _map_size.y - 1) return;
	
	cvt_spixel_to_l_theta_m(
		_spixel_list_src,
		_spixel_list_dst,
		_map_size,
		x,
		y);

}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
__global__ void Kernel_Cvt_Spixel_Similar(
	const int *				_adj_img,
	float *					_similar_img,
	const  spixel_info*		_spixel_list,
	const  gSLICr::Vector2i _adj_size)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > _adj_size.x - 1 || y > _adj_size.y - 1) return;

	cvt_spixel_similar(
				_adj_img,
				_similar_img,
				_spixel_list,
				_adj_size,
				x,
				y);

}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
seg_engine_GPU_cluster::seg_engine_GPU_cluster(const objects::settings& in_settings) : seg_engine_GPU(in_settings)
{	
	const int AdjDim = this->SpixelNum();

	const Vector2i link_size_t(AdjDim, AdjDim);

	adj_img = new IntImage(link_size_t, true, true);

	similar_img = new FloatImage(link_size_t, true, true);

	m_spixel_map_cvt = new SpixelMap(spixel_map_size, true, true);
}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
seg_engine_GPU_cluster::~seg_engine_GPU_cluster()
{
	if (adj_img != NULL) delete adj_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const dim3 gSLICr::engines::seg_engine_GPU_cluster::GetSuperPixelGridDim() const
{
	const Vector2i img_size = spixel_map->noDims;
	return GetGrid2Dim(img_size);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const dim3 gSLICr::engines::seg_engine_GPU_cluster::GetPixelGridDim() const
{
	const Vector2i img_size = source_img->noDims;
	return GetGrid2Dim(img_size);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const dim3 gSLICr::engines::seg_engine_GPU_cluster::GetGrid2Dim(Vector2i _size)  const
{
	const dim3 blockSize= GetBlockDim();
	const dim3 gridSize((int)ceil((float)_size.x / (float)blockSize.x), (int)ceil((float)_size.y / (float)blockSize.y));
	return  gridSize;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const dim3 gSLICr::engines::seg_engine_GPU_cluster::GetBlockDim() const
{
	const dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
	return blockSize;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Find_Adjacency_Matrix_E()
{
	const int* inimg_ptr = idx_img->GetData(MEMORYDEVICE_CUDA);
	int* outimg_ptr = adj_img->GetData(MEMORYDEVICE_CUDA);
	const Vector2i img_size = source_img->noDims;
	const Vector2i adj_size = adj_img->noDims;

	const dim3 blockSize	=	GetBlockDim();
	const dim3 gridSize		=	GetPixelGridDim();

	Kernel_Find_Adjacency_Matrix << <gridSize, blockSize >> >(inimg_ptr, outimg_ptr, img_size,adj_size);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Cvt_Spixel_Similar()
{
	const int*	adj_mat_ptr = adj_img->GetData(MEMORYDEVICE_CUDA);
	float*	similar_mat_ptr =similar_img->GetData(MEMORYDEVICE_CUDA);
	spixel_info* spixel_list_cvt = spixel_map->GetData(MEMORYDEVICE_CUDA);

	const Vector2i map_adj_size = adj_img->noDims;

	const dim3 blockSize	=	GetBlockDim();
	const dim3 gridSize		=	GetGrid2Dim(map_adj_size);

	Kernel_Cvt_Spixel_Similar << <gridSize, blockSize >> > (
		adj_mat_ptr,
		similar_mat_ptr,
		spixel_list_cvt,
		map_adj_size);

}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Cvt_Spixel_to_LThetaM()
{
	const spixel_info* spixel_list		=	spixel_map->GetData(MEMORYDEVICE_CUDA);
		  spixel_info* spixel_list_cvt	=	m_spixel_map_cvt->GetData(MEMORYDEVICE_CUDA);
	const Vector2i map_size = spixel_map->noDims;
	
	const dim3 blockSize = GetBlockDim();
	const dim3 gridSize =  GetSuperPixelGridDim();

	Kernel_Cvt_Spixel_to_LThetaM << <gridSize, blockSize >> > (spixel_list, spixel_list_cvt, map_size);
	
}
/*-----------------------------------------*/
/**
*
*/
/*-----------------------------------------*/
void gSLICr::engines::getMatTriangular_Float(float* _mat, const int _wh)
{

	for (size_t ri = 0; ri <_wh; ri++) {
		for (size_t ci = 0; ci < ri; ci++) {
			const int IDX = ri*_wh + ci;
			_mat[IDX] = 0;
		}
	}

}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Cvt_Similar_to_Cluster_CPU()
{
	const FloatImage * similar_img= this->Get_Similar_Matrix_Host();
	float* similar_ptr = (float*) similar_img->GetData(MEMORYDEVICE_CPU);
	const int WH = similar_img->noDims.height;
	assert(similar_img->noDims.height == similar_img->noDims.width);

	getMatTriangular_Float(similar_ptr,WH);

}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Perform_Cluster()
{
	this->Update_Cluster_Center();
	this->Find_Adjacency_Matrix_E();
	this->Cvt_Spixel_to_LThetaM();
	this->Cvt_Spixel_Similar();

	this->Cvt_Similar_to_Cluster_CPU();
	

	hipDeviceSynchronize();

}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
int gSLICr::engines::seg_engine_GPU_cluster::SpixelNum() const
{
	return this->spixel_map->noDims.x * this->spixel_map->noDims.y;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const IntImage * gSLICr::engines::seg_engine_GPU_cluster::Get_Adjacency_Matrix()
{
	adj_img->UpdateHostFromDevice();
	return adj_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const FloatImage * gSLICr::engines::seg_engine_GPU_cluster::Get_Similar_Matrix_Host()
{
	similar_img->UpdateHostFromDevice();
	return similar_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/