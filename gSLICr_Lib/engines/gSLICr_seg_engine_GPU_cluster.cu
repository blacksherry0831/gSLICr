#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of gSLICr
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_GPU.h"
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_cluster_base.h"
/*-------------------------------------------------------------------------*/
#include "gSLICr_seg_engine_GPU_cluster.h"
/*-------------------------------------------------------------------------*/
using namespace std;
using namespace gSLICr;
using namespace gSLICr::objects;
using namespace gSLICr::engines;
/*-------------------------------------------------------------------------*/
/**
*kernel function defines
*/
/*-------------------------------------------------------------------------*/
__global__ void Find_Adjacency_Matrix(
	const int * inimg,
	int * outimg,
	const  gSLICr::Vector2i img_size,
	const  gSLICr::Vector2i adj_size);
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
__global__ void Find_Adjacency_Matrix(
	const int * inimg,
	int * outimg,
	const  gSLICr::Vector2i img_size,
	const  gSLICr::Vector2i adj_size)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	const int BORDER = 1;
	if (x < img_size.x - BORDER &&
		y < img_size.y - BORDER &&
		x >= BORDER &&
		y >= BORDER) {

		find_adjacency_matrix_base(inimg, outimg, img_size, adj_size, x, y);

	}
		
}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
seg_engine_GPU_cluster::seg_engine_GPU_cluster(const objects::settings& in_settings) : seg_engine_GPU(in_settings)
{
	
	const int AdjDim = this->SpixelNum();

	const Vector2i link_size_t(AdjDim, AdjDim);

	adj_img = new IntImage(link_size_t, true, true);

}
/*----------------------------------------------------------------*/
/**
*
*/
/*----------------------------------------------------------------*/
seg_engine_GPU_cluster::~seg_engine_GPU_cluster()
{
	if (adj_img != NULL) delete adj_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Find_Adjacency_Matrix_E()
{
	const int* inimg_ptr = idx_img->GetData(MEMORYDEVICE_CUDA);
	int* outimg_ptr = adj_img->GetData(MEMORYDEVICE_CUDA);
	const Vector2i img_size = source_img->noDims;
	const Vector2i adj_size = adj_img->noDims;
	const dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
	const dim3 gridSize((int)ceil((float)img_size.x / (float)blockSize.x), (int)ceil((float)img_size.y / (float)blockSize.y));

	Find_Adjacency_Matrix << <gridSize, blockSize >> >(inimg_ptr, outimg_ptr, img_size,adj_size);
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
void gSLICr::engines::seg_engine_GPU_cluster::Perform_Cluster()
{
	this->Update_Cluster_Center();
	this->Find_Adjacency_Matrix_E();

	hipDeviceSynchronize();

}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
int gSLICr::engines::seg_engine_GPU_cluster::SpixelNum() const
{
	return this->spixel_map->noDims.x * this->spixel_map->noDims.y;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/
const IntImage * gSLICr::engines::seg_engine_GPU_cluster::Get_Adjacency_Matrix()
{
	adj_img->UpdateHostFromDevice();
	return adj_img;
}
/*-------------------------------------------------------------------------*/
/**
*
*/
/*-------------------------------------------------------------------------*/